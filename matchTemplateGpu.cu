#include "hip/hip_runtime.h"
﻿#include "matchTemplateGpu.cuh"
#include <hip/hip_runtime.h>
#include <>
#include <stdlib.h>

__global__ void matchTemplateGpu
(
    const cv::gpu::PtrStepSz<uchar> img, 
    const cv::gpu::PtrStepSz<uchar> templ, 
    cv::gpu::PtrStepSz<float> result
)
{
    const int x = blockDim.x * blockIdx.x + threadIdx.x;
    const int y = blockDim.y * blockIdx.y + threadIdx.y;

    if((x < result.cols) && (y < result.rows)){
        long sum = 0;
        for(int yy = 0; yy < templ.rows; yy++){
            for(int xx = 0; xx < templ.cols; xx++){
                int diff = (img.ptr((y+yy))[x+xx] - templ.ptr(yy)[xx]);
                sum += abs(diff);
            }
        }
        result.ptr(y)[x] = sum;
    }
}

void launchMatchTemplateGpu
(
    cv::gpu::GpuMat& img, 
    cv::gpu::GpuMat& templ, 
    cv::gpu::GpuMat& result
)
{
    cv::gpu::PtrStepSz<uchar> pImg =
        cv::gpu::PtrStepSz<uchar>(img.rows, img.cols * img.channels(), img.ptr<uchar>(), img.step);

    cv::gpu::PtrStepSz<uchar> pDst =
        cv::gpu::PtrStepSz<uchar>(templ.rows, templ.cols * templ.channels(), templ.ptr<uchar>(), templ.step);

    cv::gpu::PtrStepSz<float> pResult =
        cv::gpu::PtrStepSz<float>(result.rows, result.cols * result.channels(), result.ptr<float>(), result.step);

    const dim3 block(64, 2);
    const dim3 grid(cv::gpu::divUp(result.cols, block.x), cv::gpu::divUp(result.rows, block.y));

    matchTemplateGpu<<<grid, block>>>(pImg, pDst, pResult);

   cudaSafeCall(hipGetLastError());
   cudaSafeCall(hipDeviceSynchronize());
}

double launchMatchTemplateGpu
(
    cv::gpu::GpuMat& img, 
    cv::gpu::GpuMat& templ, 
    cv::gpu::GpuMat& result, 
    const int loop_num
)
{
    double f = 1000.0f / cv::getTickFrequency();
    int64 start = 0, end = 0;
    double time = 0.0;
    for (int i = 0; i <= loop_num; i++){
        start = cv::getTickCount();
        launchMatchTemplateGpu(img, templ, result);
        end = cv::getTickCount();
        time += (i > 0) ? ((end - start) * f) : 0;
    }
    time /= loop_num;

    return time;
}

// use static shared memory
__global__ void matchTemplateGpu_withStaticSharedMemory
(
 const cv::gpu::PtrStepSz<uchar> img,
 const cv::gpu::PtrStepSz<uchar> templ,
 cv::gpu::PtrStepSz<float> result
 )
{
  const int x = blockDim.x * blockIdx.x + threadIdx.x;
  const int y = blockDim.y * blockIdx.y + threadIdx.y;

  __shared__ uchar temp[32 * 32];

  if(threadIdx.x == 0){
    for(int yy = 0; yy < templ.rows; yy++){
      for(int xx = 0; xx < templ.cols; xx++){
	temp[yy*templ.cols+xx] = templ.ptr(yy)[xx];
      }
    }
  }
  __syncthreads();

  if((x < result.cols) && (y < result.rows)){
    long sum = 0;
    for(int yy = 0; yy < templ.rows; yy++){
      for(int xx = 0; xx < templ.cols; xx++){
	int diff = (img.ptr((y+yy))[x+xx] - temp[yy*templ.cols+xx]);
	sum += abs(diff);
      }
    }
    result.ptr(y)[x] = sum;
  }
}

// use static shared memory
void launchMatchTemplateGpu_withStaticSharedMemory
(
 cv::gpu::GpuMat& img,
 cv::gpu::GpuMat& templ,
 cv::gpu::GpuMat& result
 )
{
  cv::gpu::PtrStepSz<uchar> pImg =
    cv::gpu::PtrStepSz<uchar>(img.rows, img.cols * img.channels(), img.ptr<uchar>(), img.step);

  cv::gpu::PtrStepSz<uchar> pDst =
    cv::gpu::PtrStepSz<uchar>(templ.rows, templ.cols * templ.channels(), templ.ptr<uchar>(), templ.step);

  cv::gpu::PtrStepSz<float> pResult =
    cv::gpu::PtrStepSz<float>(result.rows, result.cols * result.channels(), result.ptr<float>(), result.step);

  const dim3 block(64, 2);
  const dim3 grid(cv::gpu::divUp(result.cols, block.x), cv::gpu::divUp(result.rows, block.y));

  matchTemplateGpu_withStaticSharedMemory<<<grid, block>>>(pImg, pDst, pResult);

  cudaSafeCall(hipGetLastError());
  cudaSafeCall(hipDeviceSynchronize());
}

// use static shared memory
double launchMatchTemplateGpu_withStaticSharedMemory
(
 cv::gpu::GpuMat& img, 
 cv::gpu::GpuMat& templ, 
 cv::gpu::GpuMat& result, 
    const int loop_num
 )
{
  double f = 1000.0f / cv::getTickFrequency();
  int64 start = 0, end = 0;
  double time = 0.0;
  for (int i = 0; i <= loop_num; i++){
    start = cv::getTickCount();
    launchMatchTemplateGpu_withStaticSharedMemory(img, templ, result);
    end = cv::getTickCount();
    time += (i > 0) ? ((end - start) * f) : 0;
  }
  time /= loop_num;

  return time;
}

// use dynamic shared memory
__global__ void matchTemplateGpu_withDynamicSharedMemory
(
 const cv::gpu::PtrStepSz<uchar> img,
 const cv::gpu::PtrStepSz<uchar> templ,
 cv::gpu::PtrStepSz<float> result
 )
{
  const int x = blockDim.x * blockIdx.x + threadIdx.x;
  const int y = blockDim.y * blockIdx.y + threadIdx.y;

  extern __shared__ uchar temp[];

  if(threadIdx.x == 0){
    for(int yy = 0; yy < templ.rows; yy++){
      for(int xx = 0; xx < templ.cols; xx++){
	temp[yy*templ.cols+xx] = templ.ptr(yy)[xx];
      }
    }
  }
  __syncthreads();

  if((x < result.cols) && (y < result.rows)){
    long sum = 0;
    for(int yy = 0; yy < templ.rows; yy++){
      for(int xx = 0; xx < templ.cols; xx++){
	int diff = (img.ptr((y+yy))[x+xx] - temp[yy*templ.cols+xx]);
	sum += abs(diff);
      }
    }
    result.ptr(y)[x] = sum;
  }
}

// use dynamic shared memory
void launchMatchTemplateGpu_withDynamicSharedMemory
(
 cv::gpu::GpuMat& img,
 cv::gpu::GpuMat& templ,
 cv::gpu::GpuMat& result
 )
{
  cv::gpu::PtrStepSz<uchar> pImg =
    cv::gpu::PtrStepSz<uchar>(img.rows, img.cols * img.channels(), img.ptr<uchar>(), img.step);

  cv::gpu::PtrStepSz<uchar> pDst =
    cv::gpu::PtrStepSz<uchar>(templ.rows, templ.cols * templ.channels(), templ.ptr<uchar>(), templ.step);

  cv::gpu::PtrStepSz<float> pResult =
    cv::gpu::PtrStepSz<float>(result.rows, result.cols * result.channels(), result.ptr<float>(), result.step);

  const dim3 block(64, 2);
  const dim3 grid(cv::gpu::divUp(result.cols, block.x), cv::gpu::divUp(result.rows, block.y));
  const size_t shared_mem_size = templ.cols*templ.rows*sizeof(uchar);

  matchTemplateGpu_withDynamicSharedMemory<<<grid, block, shared_mem_size>>>(pImg, pDst, pResult);

  cudaSafeCall(hipGetLastError());
  cudaSafeCall(hipDeviceSynchronize());
}

// use dynamic shared memory
double launchMatchTemplateGpu_withDynamicSharedMemory
(
 cv::gpu::GpuMat& img, 
 cv::gpu::GpuMat& templ, 
 cv::gpu::GpuMat& result, 
    const int loop_num
 )
{
  double f = 1000.0f / cv::getTickFrequency();
  int64 start = 0, end = 0;
  double time = 0.0;
  for (int i = 0; i <= loop_num; i++){
    start = cv::getTickCount();
    launchMatchTemplateGpu_withDynamicSharedMemory(img, templ, result);
    end = cv::getTickCount();
    time += (i > 0) ? ((end - start) * f) : 0;
  }
  time /= loop_num;

  return time;
}
