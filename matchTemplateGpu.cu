#include "hip/hip_runtime.h"
﻿#include "matchTemplateGpu.cuh"
#include <hip/hip_runtime.h>
#include <>
#include <stdlib.h>

__global__ void matchTemplateGpu
(
    const cv::gpu::PtrStepSz<uchar> img, 
    const cv::gpu::PtrStepSz<uchar> templ, 
    cv::gpu::PtrStepSz<float> result
)
{
    const int x = blockDim.x * blockIdx.x + threadIdx.x;
    const int y = blockDim.y * blockIdx.y + threadIdx.y;

    if((x < result.cols) && (y < result.rows)){
        long sum = 0;
        for(int yy = 0; yy < templ.rows; yy++){
            for(int xx = 0; xx < templ.cols; xx++){
                int diff = (img.ptr((y+yy))[x+xx] - templ.ptr(yy)[xx]);
                sum += abs(diff);
            }
        }
        result.ptr(y)[x] = sum;
    }
}

void launchMatchTemplateGpu
(
    cv::gpu::GpuMat& img, 
    cv::gpu::GpuMat& templ, 
    cv::gpu::GpuMat& result,
    const dim3 block
)
{
    cv::gpu::PtrStepSz<uchar> pImg =
        cv::gpu::PtrStepSz<uchar>(img.rows, img.cols * img.channels(), img.ptr<uchar>(), img.step);

    cv::gpu::PtrStepSz<uchar> pDst =
        cv::gpu::PtrStepSz<uchar>(templ.rows, templ.cols * templ.channels(), templ.ptr<uchar>(), templ.step);

    cv::gpu::PtrStepSz<float> pResult =
        cv::gpu::PtrStepSz<float>(result.rows, result.cols * result.channels(), result.ptr<float>(), result.step);

    // const dim3 block = blockSize;
    const dim3 grid(cv::gpu::divUp(result.cols, block.x), cv::gpu::divUp(result.rows, block.y));

    matchTemplateGpu<<<grid, block>>>(pImg, pDst, pResult);

   cudaSafeCall(hipGetLastError());
   cudaSafeCall(hipDeviceSynchronize());
}

double launchMatchTemplateGpu
(
    cv::gpu::GpuMat& img, 
    cv::gpu::GpuMat& templ, 
    cv::gpu::GpuMat& result, 
    const dim3 block,
    const int loop_num
)
{
    double f = 1000.0f / cv::getTickFrequency();
    int64 start = 0, end = 0;
    double time = 0.0;
    for (int i = 0; i <= loop_num; i++){
        start = cv::getTickCount();
        launchMatchTemplateGpu(img, templ, result, block);
        end = cv::getTickCount();
        time += (i > 0) ? ((end - start) * f) : 0;
    }
    time /= loop_num;

    return time;
}

// use static shared memory
__global__ void matchTemplateGpu_withStaticSharedMemory
(
 const cv::gpu::PtrStepSz<uchar> img,
 const cv::gpu::PtrStepSz<uchar> templ,
 cv::gpu::PtrStepSz<float> result
 )
{
  const int x = blockDim.x * blockIdx.x + threadIdx.x;
  const int y = blockDim.y * blockIdx.y + threadIdx.y;

  __shared__ uchar temp[32 * 32];

  if(threadIdx.x == 0){
    for(int yy = 0; yy < templ.rows; yy++){
      for(int xx = 0; xx < templ.cols; xx++){
	temp[yy*templ.cols+xx] = templ.ptr(yy)[xx];
      }
    }
  }
  __syncthreads();

  if((x < result.cols) && (y < result.rows)){
    long sum = 0;
    for(int yy = 0; yy < templ.rows; yy++){
      for(int xx = 0; xx < templ.cols; xx++){
	int diff = (img.ptr((y+yy))[x+xx] - temp[yy*templ.cols+xx]);
	sum += abs(diff);
      }
    }
    result.ptr(y)[x] = sum;
  }
}


// use static shared memory
void launchMatchTemplateGpu_withStaticSharedMemory
(
 cv::gpu::GpuMat& img,
 cv::gpu::GpuMat& templ,
 cv::gpu::GpuMat& result,
 const dim3 block
 )
{
  cv::gpu::PtrStepSz<uchar> pImg =
    cv::gpu::PtrStepSz<uchar>(img.rows, img.cols * img.channels(), img.ptr<uchar>(), img.step);

  cv::gpu::PtrStepSz<uchar> pDst =
    cv::gpu::PtrStepSz<uchar>(templ.rows, templ.cols * templ.channels(), templ.ptr<uchar>(), templ.step);

  cv::gpu::PtrStepSz<float> pResult =
    cv::gpu::PtrStepSz<float>(result.rows, result.cols * result.channels(), result.ptr<float>(), result.step);

  // const dim3 block(64, 2);
  const dim3 grid(cv::gpu::divUp(result.cols, block.x), cv::gpu::divUp(result.rows, block.y));

  matchTemplateGpu_withStaticSharedMemory<<<grid, block>>>(pImg, pDst, pResult);

  cudaSafeCall(hipGetLastError());
  cudaSafeCall(hipDeviceSynchronize());
}

// use static shared memory
double launchMatchTemplateGpu_withStaticSharedMemory
(
 cv::gpu::GpuMat& img, 
 cv::gpu::GpuMat& templ, 
 cv::gpu::GpuMat& result, 
 const dim3 block,
 const int loop_num
 )
{
  double f = 1000.0f / cv::getTickFrequency();
  int64 start = 0, end = 0;
  double time = 0.0;
  for (int i = 0; i <= loop_num; i++){
    start = cv::getTickCount();
    launchMatchTemplateGpu_withStaticSharedMemory(img, templ, result, block);
    end = cv::getTickCount();
    time += (i > 0) ? ((end - start) * f) : 0;
  }
  time /= loop_num;

  return time;
}

// use dynamic shared memory
__global__ void matchTemplateGpu_withDynamicSharedMemory
(
 const cv::gpu::PtrStepSz<uchar> img,
 const cv::gpu::PtrStepSz<uchar> templ,
 cv::gpu::PtrStepSz<float> result
 )
{
  const int x = blockDim.x * blockIdx.x + threadIdx.x;
  const int y = blockDim.y * blockIdx.y + threadIdx.y;

  extern __shared__ uchar temp[];

  if(threadIdx.x == 0){
    for(int yy = 0; yy < templ.rows; yy++){
      for(int xx = 0; xx < templ.cols; xx++){
	temp[yy*templ.cols+xx] = templ.ptr(yy)[xx];
      }
    }
  }
  __syncthreads();

  if((x < result.cols) && (y < result.rows)){
    long sum = 0;
    for(int yy = 0; yy < templ.rows; yy++){
      for(int xx = 0; xx < templ.cols; xx++){
	int diff = (img.ptr((y+yy))[x+xx] - temp[yy*templ.cols+xx]);
	sum += abs(diff);
      }
    }
    result.ptr(y)[x] = sum;
  }
}

// use dynamic shared memory
void launchMatchTemplateGpu_withDynamicSharedMemory
(
 cv::gpu::GpuMat& img,
 cv::gpu::GpuMat& templ,
 cv::gpu::GpuMat& result,
 const dim3 block
 )
{
  cv::gpu::PtrStepSz<uchar> pImg =
    cv::gpu::PtrStepSz<uchar>(img.rows, img.cols * img.channels(), img.ptr<uchar>(), img.step);

  cv::gpu::PtrStepSz<uchar> pDst =
    cv::gpu::PtrStepSz<uchar>(templ.rows, templ.cols * templ.channels(), templ.ptr<uchar>(), templ.step);

  cv::gpu::PtrStepSz<float> pResult =
    cv::gpu::PtrStepSz<float>(result.rows, result.cols * result.channels(), result.ptr<float>(), result.step);

  // const dim3 block(64, 2);
  const dim3 grid(cv::gpu::divUp(result.cols, block.x), cv::gpu::divUp(result.rows, block.y));
  const size_t shared_mem_size = templ.cols*templ.rows*sizeof(uchar);

  matchTemplateGpu_withDynamicSharedMemory<<<grid, block, shared_mem_size>>>(pImg, pDst, pResult);

  cudaSafeCall(hipGetLastError());
  cudaSafeCall(hipDeviceSynchronize());
}


// use dynamic shared memory
double launchMatchTemplateGpu_withDynamicSharedMemory
(
 cv::gpu::GpuMat& img, 
 cv::gpu::GpuMat& templ, 
 cv::gpu::GpuMat& result, 
 const dim3 block,
 const int loop_num
 )
{
  double f = 1000.0f / cv::getTickFrequency();
  int64 start = 0, end = 0;
  double time = 0.0;
  for (int i = 0; i <= loop_num; i++){
    start = cv::getTickCount();
    launchMatchTemplateGpu_withDynamicSharedMemory(img, templ, result, block);
    end = cv::getTickCount();
    time += (i > 0) ? ((end - start) * f) : 0;
  }
  time /= loop_num;

  return time;
}

// use static shared memory with loop unrolling
__global__ void matchTemplateGpu_withStaticSharedMemory_withLoopUnrolling
(
 const cv::gpu::PtrStepSz<uchar> img,
 const cv::gpu::PtrStepSz<uchar> templ,
 cv::gpu::PtrStepSz<float> result
 )
{
  const int x = blockDim.x * blockIdx.x + threadIdx.x;
  const int y = blockDim.y * blockIdx.y + threadIdx.y;

  __shared__ uchar temp[32 * 32];

  if(threadIdx.x == 0){
    for(int yy = 0; yy < templ.rows; yy++){
      #pragma unroll 4
      for(int xx = 0; xx < templ.cols; xx += 4){
	temp[yy*templ.cols+xx] = templ.ptr(yy)[xx];
	temp[yy*templ.cols+xx+1] = templ.ptr(yy)[xx+1];
	temp[yy*templ.cols+xx+2] = templ.ptr(yy)[xx+2];
	temp[yy*templ.cols+xx+3] = templ.ptr(yy)[xx+3];
      }
    }
  }
  __syncthreads();

  if((x < result.cols) && (y < result.rows)){
    long sum = 0;
    for(int yy = 0; yy < templ.rows; yy++){
      #pragma unroll 4
      for(int xx = 0; xx < templ.cols; xx += 4){
	int diff = abs(img.ptr((y+yy))[x+xx] - temp[yy*templ.cols+xx]);
	diff += abs(img.ptr((y+yy))[x+xx+1] - temp[yy*templ.cols+xx+1]);
	diff += abs(img.ptr((y+yy))[x+xx+2] - temp[yy*templ.cols+xx+2]);
	diff += abs(img.ptr((y+yy))[x+xx+3] - temp[yy*templ.cols+xx+3]);
	sum += diff;
      }
    }
    result.ptr(y)[x] = sum;
  }
}


// use static shared memory with loop unrolling
void launchMatchTemplateGpu_withStaticSharedMemory_withLoopUnrolling
(
 cv::gpu::GpuMat& img,
 cv::gpu::GpuMat& templ,
 cv::gpu::GpuMat& result,
 const dim3 block
 )
{
  cv::gpu::PtrStepSz<uchar> pImg =
    cv::gpu::PtrStepSz<uchar>(img.rows, img.cols * img.channels(), img.ptr<uchar>(), img.step);

  cv::gpu::PtrStepSz<uchar> pDst =
    cv::gpu::PtrStepSz<uchar>(templ.rows, templ.cols * templ.channels(), templ.ptr<uchar>(), templ.step);

  cv::gpu::PtrStepSz<float> pResult =
    cv::gpu::PtrStepSz<float>(result.rows, result.cols * result.channels(), result.ptr<float>(), result.step);

  // const dim3 block(64, 2);
  const dim3 grid(cv::gpu::divUp(result.cols, block.x), cv::gpu::divUp(result.rows, block.y));

  matchTemplateGpu_withStaticSharedMemory_withLoopUnrolling<<<grid, block>>>(pImg, pDst, pResult);

  cudaSafeCall(hipGetLastError());
  cudaSafeCall(hipDeviceSynchronize());
}

// use static shared memory with loop unrolling
double launchMatchTemplateGpu_withStaticSharedMemory_withLoopUnrolling
(
 cv::gpu::GpuMat& img, 
 cv::gpu::GpuMat& templ, 
 cv::gpu::GpuMat& result, 
 const dim3 block,
 const int loop_num
 )
{
  double f = 1000.0f / cv::getTickFrequency();
  int64 start = 0, end = 0;
  double time = 0.0;
  for (int i = 0; i <= loop_num; i++){
    start = cv::getTickCount();
    launchMatchTemplateGpu_withStaticSharedMemory_withLoopUnrolling(img, templ, result, block);
    end = cv::getTickCount();
    time += (i > 0) ? ((end - start) * f) : 0;
  }
  time /= loop_num;

  return time;
}

// use dynamic shared memory with loop unrolling
__global__ void matchTemplateGpu_withDynamicSharedMemory_withLoopUnrolling
(
 const cv::gpu::PtrStepSz<uchar> img,
 const cv::gpu::PtrStepSz<uchar> templ,
 cv::gpu::PtrStepSz<float> result
 )
{
  const int x = blockDim.x * blockIdx.x + threadIdx.x;
  const int y = blockDim.y * blockIdx.y + threadIdx.y;

  extern __shared__ uchar temp[];

  if(threadIdx.x == 0){
    for(int yy = 0; yy < templ.rows; yy++){
      #pragma unroll 4
      for(int xx = 0; xx < templ.cols; xx += 4){
	temp[yy*templ.cols+xx] = templ.ptr(yy)[xx];
	temp[yy*templ.cols+xx+1] = templ.ptr(yy)[xx+1];
	temp[yy*templ.cols+xx+2] = templ.ptr(yy)[xx+2];
	temp[yy*templ.cols+xx+3] = templ.ptr(yy)[xx+3];
      }
    }
  }
  __syncthreads();

  if((x < result.cols) && (y < result.rows)){
    long sum = 0;
    for(int yy = 0; yy < templ.rows; yy++){
      #pragma unroll 4
      for(int xx = 0; xx < templ.cols; xx += 4){
	int diff = abs(img.ptr((y+yy))[x+xx] - temp[yy*templ.cols+xx]);
	diff += abs(img.ptr((y+yy))[x+xx+1] - temp[yy*templ.cols+xx+1]);
	diff += abs(img.ptr((y+yy))[x+xx+2] - temp[yy*templ.cols+xx+2]);
	diff += abs(img.ptr((y+yy))[x+xx+3] - temp[yy*templ.cols+xx+3]);
	sum += diff;
      }
    }
    result.ptr(y)[x] = sum;
  }
}

// use dynamic shared memory
void launchMatchTemplateGpu_withDynamicSharedMemory_withLoopUnrolling
(
 cv::gpu::GpuMat& img,
 cv::gpu::GpuMat& templ,
 cv::gpu::GpuMat& result,
 const dim3 block
 )
{
  cv::gpu::PtrStepSz<uchar> pImg =
    cv::gpu::PtrStepSz<uchar>(img.rows, img.cols * img.channels(), img.ptr<uchar>(), img.step);

  cv::gpu::PtrStepSz<uchar> pDst =
    cv::gpu::PtrStepSz<uchar>(templ.rows, templ.cols * templ.channels(), templ.ptr<uchar>(), templ.step);

  cv::gpu::PtrStepSz<float> pResult =
    cv::gpu::PtrStepSz<float>(result.rows, result.cols * result.channels(), result.ptr<float>(), result.step);

  // const dim3 block(64, 2);
  const dim3 grid(cv::gpu::divUp(result.cols, block.x), cv::gpu::divUp(result.rows, block.y));
  const size_t shared_mem_size = templ.cols*templ.rows*sizeof(uchar);

  matchTemplateGpu_withDynamicSharedMemory_withLoopUnrolling<<<grid, block, shared_mem_size>>>(pImg, pDst, pResult);

  cudaSafeCall(hipGetLastError());
  cudaSafeCall(hipDeviceSynchronize());
}


// use dynamic shared memory
double launchMatchTemplateGpu_withDynamicSharedMemory_withLoopUnrolling
(
 cv::gpu::GpuMat& img, 
 cv::gpu::GpuMat& templ, 
 cv::gpu::GpuMat& result, 
 const dim3 block,
 const int loop_num
 )
{
  double f = 1000.0f / cv::getTickFrequency();
  int64 start = 0, end = 0;
  double time = 0.0;
  for (int i = 0; i <= loop_num; i++){
    start = cv::getTickCount();
    launchMatchTemplateGpu_withDynamicSharedMemory_withLoopUnrolling(img, templ, result, block);
    end = cv::getTickCount();
    time += (i > 0) ? ((end - start) * f) : 0;
  }
  time /= loop_num;

  return time;
}
